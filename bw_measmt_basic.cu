#include "hip/hip_runtime.h"
// results on A5000 (Nblocks=256*4096, Nthreads=256) =>
// single write: Equivalent mem transfer speed: 709.4GB/s
// read + write: Equivalent mem transfer speed: 685.4GB/s
// 2 reads + write: Equivalent mem transfer speed: 692.0GB/s

#include <stdio.h>
#include <iostream>

__global__ void hello_world()
{
  printf("GPU hello world!\n");
}

__global__ void vector_add(float *out, float *a, float *b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("i = %d\n", i);
    out[i] = a[i] + b[i];
    // out[i] = 1.0;
    // out[i] = a[i];
}

int main() {
    
    int nDevices;
    hipGetDeviceCount(&nDevices);
    
    printf("Number of devices: %d\n", nDevices);
    
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
            prop.memoryClockRate/1024);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
    }

    int Nblocks = 256*4096;
    int Nthreads = 256;
    int N = Nblocks * Nthreads;

    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    float *d_a, *d_b, *d_out;

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vector_add<<<Nblocks,Nthreads>>>(d_out, d_a, d_b);
    // hello_world<<<1,4>>>();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipDeviceSynchronize();
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time elapsed: %.1f ms\n", milliseconds);
    printf("Equivalent mem transfer speed: %.1fGB/s\n", 
        3.0*4.0*(float)N/(1'000'000.0*milliseconds));
    printf("sizeof(float): %d\n", (int)sizeof(float));
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);

    return 0;
}
